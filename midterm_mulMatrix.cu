/*
./mulMatrix 
./mulMatrix Starting...
Using Device 0: NVIDIA GeForce RTX 3050 Ti Laptop GPU
Matrix size: nx 256 ny 256
nxy=65536
sumMatrixOnHost elapsed   35.49 ms
sumMatrixOnGPU_2D1D_v1 <<<(1024,1), (1,256)>>> elapsed  20.046 ms
Results match.
sumMatrixOnGPU_1D1D_v1 <<<(64,1), (1024,1)>>> elapsed   2.542 ms
Results match.
sumMatrixOnGPU_2D2D_v1 <<<(8,8), (32,32)>>> elapsed   0.089 ms
Results match.

*/

#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;
    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }
        ia += nx;
        ib += nx;
        ic += nx;
    }
}

// mul
void multiplyMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            float sum = 0.0f;
            for (int k = 0; k < nx; k++) {
                sum += A[iy * nx + k] * B[k * nx + ix];
            }
            C[iy * nx + ix] = sum;
        }
    }
}


__global__ void sumMatrixOnGPU_1D1D_v1(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    MatC[index] = MatA[index] + MatB[index];
}

__global__ void sumMatrixOnGPU_1D1D_v2(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx)
    {
        for (int iy = 0; iy < ny; iy++)
        {
            int idx = iy * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}

// mul
__global__ void multiplyMatrixOnGPU_1D1D(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx) {
        for (int iy = 0; iy < ny; iy++) {
            float sum = 0.0f;
            for (int k = 0; k < nx; k++) {
                sum += MatA[iy * nx + k] * MatB[k * nx + ix];
            }
            MatC[iy * nx + ix] = sum;
        }
    }
}



/* Practice: 新增2D grids of 1D block的v1與v2函式及相關的對應程式碼 */
__global__ void sumMatrixOnGPU_2DGrid1DBlock_v1(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < nx) {
        for (int iy = 0; iy < ny; iy++) {
            int idx = iy * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}

// 這個版本將使用類似的配置，但會對每個線程使用一個更嚴格的界限檢查，確保不會有任何越界錯誤。
__global__ void sumMatrixOnGPU_2DGrid1DBlock_v2(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= nx) return;  // Ensure ix is within bounds before proceeding.

    for (int iy = 0; iy < ny; iy++) {
        int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}
/**/

// mul
__global__ void multiplyMatrixOnGPU_2DGrid1DBlock(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < nx) {
        for (int iy = 0; iy < ny; iy++) {
            float sum = 0.0f;
            for (int k = 0; k < nx; k++) {
                sum += MatA[iy * nx + k] * MatB[k * nx + ix];
            }
            MatC[iy * nx + ix] = sum;
        }
    }
}

__global__ void sumMatrixOnGPU_2D2D_v1(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int bIdx = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned int index = bIdx * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    // unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    // unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // if (ix < nx && iy < ny)
    MatC[index] = MatA[index] + MatB[index];
}

__global__ void sumMatrixOnGPU_2D2D_v2(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}

// TODO: main
__global__ void calculateMatrix2D2D(float *MatA, float *MatB, float *MatC, float *MatD, float *MatI, float *MatOut, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatI[idx] + MatB[idx];

        float sum = 0.0f;
        for (int k = 0; k < nx; k++) {
            sum += MatA[iy * nx + k] * MatC[k * nx + ix];
        }
        MatD[idx] = sum;

        unsigned int idx_in = iy * nx + ix;
        unsigned int idx_out = ix * ny + iy;
        MatOut[idx_out] = MatD[idx_in];
    }
}

// mul
__global__ void multiplyMatrixOnGPU_2D2D(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx && iy < ny) {
        float sum = 0.0f;
        for (int k = 0; k < nx; k++) {
            sum += MatA[iy * nx + k] * MatB[k * nx + ix];
        }
        MatC[iy * nx + ix] = sum;
    }
}


void initialData(float *ip, unsigned int size)
{
    // time_t t;
    int index = 1;
    // srand((unsigned int)time(&t));
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
        // ip[i] = (float)(rand() & 0xFF) / 10.0f;''
            ip[i * size + j] = index % 100;
            index += 1;
            //printf("aaaaaa%d\n", i * size + j);
        }
        //row++;
        //col++;
    }
}

// TODO: initialData_I
void initialData_I(float *ip, unsigned int size)
{
    // time_t t;
    // srand((unsigned int)time(&t));
    int row = 0;
    int col = 0;
    for (unsigned int i = 0; i < size; i++) {
        // ip[i] = (float)(rand() & 0xFF) / 10.0f;
        ip[row * size + col] = 1;
        row++;
        col++;

        // printf("aaaaaa%d\n", row);
    }
    // printf("aaa\n\n\n");
    
}

// TODO: printMatrix
void printMatrix(float *C, const int nx, const int ny)
{
    if(nx > 5) return;
    float *ic = C;
    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            printf("%5.2f ", ic[ix]);
        }
        ic += nx;
        printf("\n");
    }
    printf("\n");
}


void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-2; 
    int match = 1;
    for (int i = 0; i < N; i++) {
        if (fabs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Results do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) {
        printf("Results match.\n");
    }
    return;
}

double cpuSecond()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// TODO: transposeMatrix
__global__ void transposeMatrixGPU(float *MatIn, float *MatOut, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (ix < nx && iy < ny)
    {
        unsigned int idx_in = iy * nx + ix;
        unsigned int idx_out = ix * ny + iy;
        MatOut[idx_out] = MatIn[idx_in];
    }
}

void transposeMatrix(float *MatIn, float *MatOut, int nx, int ny)
{
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++) {
            int idx_in = iy * nx + ix;
            int idx_out = ix * ny + iy;
            MatOut[idx_out] = MatIn[idx_in];
        }
    }

}

int main(int argc, char **argv)
{
    int dev = 0;
    double iStart = 0, iElaps = 0;
    hipDeviceProp_t deviceProp;
    hipEvent_t time1, time2;
    float kernelExecutionTime;
    printf("%s Starting...\n", argv[0]);
    
    // set up device
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    // 使用者資料維度為nx * ny
    // int nx = 1 << 10;
    // int ny = 1 << 10;
    int nx = 3000;
    int ny = 3000;
    // int nx = 3;
    // int ny = 3;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);
    float *h_A, *h_B, *h_C, *h_I, *hostRef, *hostRef2, *hostRef3, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);

    h_I = (float *)malloc(nBytes);

    hostRef = (float *)malloc(nBytes);
    hostRef2 = (float *)malloc(nBytes);
    hostRef3 = (float *)malloc(nBytes);

    gpuRef = (float *)malloc(nBytes);

    // 初始化使用者資料
    printf("nxy=%d\n", nxy);
    initialData(h_A, nx);
    initialData(h_B, nx);
    initialData(h_C, nx);

    printf("A\n");
    printMatrix(h_A, nx, ny);
    printf("B\n");
    printMatrix(h_B, nx, ny);

    printf("I\n");
    initialData_I(h_I, nx);
    printMatrix(h_I, nx, ny);

    memset(hostRef, 0, nBytes);
    memset(hostRef2, 0, nBytes);
    memset(hostRef3, 0, nBytes);

    memset(gpuRef, 0, nBytes);

    iStart = cpuSecond();
    // 執行CPU矩陣相加函式
    // sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    // TODO: sumMatrixOnHost
    sumMatrixOnHost(h_I, h_B, hostRef, nx, ny);
    printf("I+B\n");
    printMatrix(hostRef, nx, ny);

    multiplyMatrixOnHost(h_A, hostRef, hostRef2, nx, ny);
    printf("A*(I+B)\n");
    printMatrix(hostRef2, nx, ny);

    transposeMatrix(hostRef2, hostRef3, nx, ny);
    printf("Transpose Matrix\n");
    printMatrix(hostRef3, nx, ny);


    iElaps = cpuSecond() - iStart;
    printf("sumMatrixOnHost elapsed %7.2f ms\n", iElaps * 1000);
    float *d_MatA, *d_MatB, *d_MatC, *d_MatI, *d_MatD, *d_MatE;
    hipMalloc((void **)&d_MatA, nBytes);
    hipMalloc((void **)&d_MatB, nBytes);
    hipMalloc((void **)&d_MatC, nBytes);
    hipMalloc((void **)&d_MatD, nBytes);
    hipMalloc((void **)&d_MatE, nBytes);
    hipMalloc((void **)&d_MatI, nBytes);

    // transfer data from host to device
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatI, h_I, nBytes, hipMemcpyHostToDevice);

    int dimx11v1 = 1024;
    int dimy11v1 = 1;
    dim3 block11v1(dimx11v1, dimy11v1);
    dim3 grid11v1((nx * ny + block11v1.x - 1) / block11v1.x);

    // TODO: use this
    int dimx11v2 = 1024;
    int dimy11v2 = 1;
    dim3 block11v2(dimx11v2, dimy11v2);
    dim3 grid11v2((nx + block11v2.x - 1) / block11v2.x);

    int dimx22v1 = 32;
    int dimy22v1 = 32;
    dim3 block22v1(dimx22v1, dimy22v1);
    dim3 grid22v1((nx + block22v1.x - 1) / block22v1.x, (ny + block22v1.y - 1) / block22v1.y);
    int dimx22v2 = 32;
    int dimy22v2 = 32;
    dim3 block22v2(dimx22v2, dimy22v2);
    dim3 grid22v2((nx + block22v2.x - 1) / block22v2.x, (ny + block22v2.y - 1) / block22v2.y);
    hipEventCreate(&time1);
    hipEventCreate(&time2);

    // // 2D 1D v1
    // int threadsPerBlock = 1024;
    // dim3 block2D1Dv1(threadsPerBlock);
    // dim3 grid2D1Dv1((nx + block2D1Dv1.x - 1) / block2D1Dv1.x, ny);

    // // 2D Grids of 1D Blocks v1
    // cudaEventRecord(time1, 0);
    // // sumMatrixOnGPU_2DGrid1DBlock_v1<<<grid2D1Dv1, block2D1Dv1>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // multiplyMatrixOnGPU_2DGrid1DBlock<<<grid2D1Dv1, block2D1Dv1>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // cudaEventRecord(time2, 0);

    // cudaEventSynchronize(time1);
    // cudaEventSynchronize(time2);
    // cudaEventElapsedTime(&kernelExecutionTime, time1, time2);

    // printf("sumMatrixOnGPU_2D1D_v1 <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n",
    //        block2D1Dv1.x, block2D1Dv1.y, grid2D1Dv1.x, grid2D1Dv1.y, kernelExecutionTime);

    // cudaMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviceToHost);

    // // check device results
    // checkResult(hostRef, gpuRef, nxy);


    // // 2D 1D v2
    // threadsPerBlock = 1024;
    // dim3 block2D1Dv2(threadsPerBlock);
    // dim3 grid2D1Dv2((nx + block2D1Dv2.x - 1) / block2D1Dv2.x, ny);

    // // 2D Grids of 1D Blocks v2
    // cudaEventRecord(time1, 0);
    // sumMatrixOnGPU_2DGrid1DBlock_v2<<<grid2D1Dv2, block2D1Dv2>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // cudaEventRecord(time2, 0);

    // cudaEventSynchronize(time1);
    // cudaEventSynchronize(time2);
    // cudaEventElapsedTime(&kernelExecutionTime, time1, time2);

    // printf("sumMatrixOnGPU_2D1D_v2 <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n",
    //        block2D1Dv2.x, block2D1Dv2.y, grid2D1Dv2.x, grid2D1Dv2.y, kernelExecutionTime);

    // cudaMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviceToHost);

    // // check device results
    // checkResult(hostRef, gpuRef, nxy);

    // // 1D-1D v1
    // cudaEventRecord(time1, 0);
    // // sumMatrixOnGPU_1D1D_v1<<<grid11v1, block11v1>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // multiplyMatrixOnGPU_1D1D<<<grid11v1, block11v1>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // cudaEventRecord(time2, 0);
    // cudaEventSynchronize(time1);
    // cudaEventSynchronize(time2);
    // cudaEventElapsedTime(&kernelExecutionTime, time1, time2);
    // printf("sumMatrixOnGPU_1D1D_v1 <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n", grid11v1.x,
    //        grid11v1.y, block11v1.x, block11v1.y, kernelExecutionTime);
    // cudaMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviceToHost);

    // // check device results
    // checkResult(hostRef, gpuRef, nxy);

    // // 1D-1D v2
    // cudaEventRecord(time1, 0);
    // sumMatrixOnGPU_1D1D_v2<<<grid11v2, block11v2>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // cudaEventRecord(time2, 0);
    // cudaEventSynchronize(time1);
    // cudaEventSynchronize(time2);
    // cudaEventElapsedTime(&kernelExecutionTime, time1, time2);
    // printf("sumMatrixOnGPU_1D1D_v2 <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n",
    //        grid11v2.x, grid11v2.y, block11v2.x, block11v2.y, kernelExecutionTime);
    // cudaMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviceToHost);
    // checkResult(hostRef, gpuRef, nxy);

    // 2D-2D v1
    hipEventRecord(time1, 0);
    sumMatrixOnGPU_2D2D_v1<<<grid22v1, block22v1>>>(d_MatB, d_MatI, d_MatC, nx, ny);
    multiplyMatrixOnGPU_2D2D<<<grid22v1, block22v1>>>(d_MatA, d_MatC, d_MatD, nx, ny);
    transposeMatrixGPU<<<grid22v1, block22v1>>>(d_MatD, d_MatE, nx, ny);
    hipEventRecord(time2, 0);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime, time1, time2);

    printf("sumMatrixOnGPU_2D2D_v1 <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n",
           grid22v1.x, grid22v1.y, block22v1.x, block22v1.y, kernelExecutionTime);
    hipMemcpy(gpuRef, d_MatE, nBytes, hipMemcpyDeviceToHost);
    checkResult(hostRef3, gpuRef, nxy);

    // cudaMemcpy(h_C, d_MatE, nBytes, cudaMemcpyDeviceToHost);
    printf("2D 2D\n");
    printMatrix(gpuRef, nx, ny);

    
    //TODO: all in one
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_MatI, h_I, nBytes, hipMemcpyHostToDevice);

    hipEventRecord(time1, 0);
    calculateMatrix2D2D<<<grid22v1, block22v1>>>(d_MatA, d_MatB, d_MatC, d_MatD, d_MatI, d_MatE, nx, ny);
    hipEventRecord(time2, 0);

    hipEventSynchronize(time1);
    hipEventSynchronize(time2);
    hipEventElapsedTime(&kernelExecutionTime, time1, time2);

    printf("2D 2D All In One <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n",
           grid22v1.x, grid22v1.y, block22v1.x, block22v1.y, kernelExecutionTime);
    hipMemcpy(gpuRef, d_MatE, nBytes, hipMemcpyDeviceToHost);
    checkResult(hostRef3, gpuRef, nxy);

    printf("2D 2D All In One\n");
    printMatrix(gpuRef, nx, ny);

    // // 2D-2D v2
    // cudaEventRecord(time1, 0);
    // sumMatrixOnGPU_2D2D_v2<<<grid22v2, block22v2>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    // cudaEventRecord(time2, 0);
    // cudaEventSynchronize(time1);
    // cudaEventSynchronize(time2);
    // cudaEventElapsedTime(&kernelExecutionTime, time1, time2);
    // printf("sumMatrixOnGPU_2D2D_v2 <<<(%d,%d), (%d,%d)>>> elapsed %7.3f ms\n",
    //        grid22v2.x, grid22v2.y, block22v2.x, block22v2.y, kernelExecutionTime);
    // cudaMemcpy(gpuRef, d_MatC, nBytes, cudaMemcpyDeviceToHost);
    // checkResult(hostRef, gpuRef, nxy);

    hipFree(d_MatA);
    hipFree(d_MatB);
    hipFree(d_MatC);
    hipFree(d_MatD);
    hipFree(d_MatE);
    hipFree(d_MatI);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_I);

    free(hostRef);
    free(hostRef2);
    free(hostRef3);

    free(gpuRef);
    hipDeviceReset();
    return (0);
}